#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "cudaFractal.h"
#include "utils.h"

#define NUM_THREADS_PER_BLOCK 256

__global__ void
drawFractal(char *out, double center_x, double center_y, double w_real, double h_real, int w_image, int h_image,
            int max_iter) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < h_image * w_image) {
        int image_x = i % w_image;
        int image_y = i / w_image;
        double c_x = fma((double)image_x, w_real / w_image, center_x - w_real / 2.0);
        double c_y = fma((double)(h_image - image_y), h_real / h_image, center_y - h_real / 2.0);

        float iter = 0;
        double z_x = 0;
        double z_y = 0;
        while (iter < max_iter && (z_x * z_x + z_y * z_y) < 4) {
            iter++;
            double tmp = z_x;
            z_x = z_x * z_x - z_y * z_y + c_x;
            z_y = 2 * z_y * tmp + c_y;
        }

        out[i] = (char) (iter * 255 / max_iter);
    }
}

char *create_fractal() {
    std::cout << "center : " << center_x << " " << center_y << std::endl;
    std::cout << "dimension : " << r_width << " " << r_height << std::endl;
    std::cout << "max iteration : " << max_iter << std::endl;

    //calling the kernel !
    drawFractal << < nb_block, nb_threads >> > (deviceImage, center_x, center_y, r_width, r_height, width, height, max_iter);

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    size_t size = height * width * sizeof(char);
    HANDLE_ERROR(hipMemcpy(hostImage, deviceImage, size, hipMemcpyDeviceToHost));

    return hostImage;
}

void init_gpu(int w, int h) {
    height = h;
    width = w;

    r_height = 2.5;
    r_width = r_height * (double) width / (double) height;

    std::cout << "dimension " << r_width << " " << r_height << std::endl;

    size_t size = height * width * sizeof(char);

    hostImage = (char *) malloc(size);
    HANDLE_ERROR(hipMalloc(&deviceImage, size));

    nb_block = ceil((double) (height * width) / (double) NUM_THREADS_PER_BLOCK);
    nb_threads = NUM_THREADS_PER_BLOCK;
}

void delete_gpu() {
    hipFree(deviceImage);
    free(hostImage);
}

void set_center(int pos_x, int pos_y) {
    center_x = ((double) pos_x / (double) width * r_width) + center_x - r_width / 2.0;
    center_y = ((double) (height - pos_y) / (double) height * r_height) + center_y - r_height / 2.0;
}

void set_zoom_scale(double scale) {
    r_width *= scale;
    r_height *= scale;

    max_iter += 20 * ((scale < 1) ? 1 : -1);
}
