#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

#include "cudaFractal.h"
#include "utils.h"

#define NUM_THREADS_PER_BLOCK 256
#define NUM_COLOR 3

#define SATURATION 1.F
#define VALUE 0.8F

__global__ void
drawFractal(char *out, double center_x, double center_y, double w_real, double h_real, int w_image, int h_image,
            int max_iter) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < h_image * w_image) {
        int image_x = i % w_image;
        int image_y = i / w_image;
        double c_x = fma((double) image_x, w_real / w_image, center_x - w_real / 2.0);
        double c_y = fma((double) (h_image - image_y), h_real / h_image, center_y - h_real / 2.0);

        float iter = 0;
        double z_x = 0;
        double z_y = 0;
        while (iter < max_iter && (z_x * z_x + z_y * z_y) < 4) {
            iter++;
            double tmp = z_x;
            z_x = z_x * z_x - z_y * z_y + c_x;
            z_y = 2 * z_y * tmp + c_y;
        }

        int h = (int) (iter * 240 / max_iter);
        int h_ = (h / 60) % 6;
        float f = ((float) h / 60.F) - (float) h_;
        float l = VALUE * (1.F - SATURATION);
        float m = VALUE * (1.F - f * SATURATION);
        float n = VALUE * (1.F - (1.F - f) * SATURATION);

        int color_index = i * 3;

        switch (h_) {
            case 0:
                out[color_index] = (char) (VALUE * 255.F);
                out[color_index + 1] = (char) (n * 255.F);
                out[color_index + 2] = (char) (l * 255.F);
                break;
            case 1:
                out[color_index] = (char) (m * 255.F);
                out[color_index + 1] = (char) (VALUE * 255.F);
                out[color_index + 2] = (char) (l * 255.F);
                break;
            case 2:
                out[color_index] = (char) (l * 255.F);
                out[color_index + 1] = (char) (VALUE * 255.F);
                out[color_index + 2] = (char) (n * 255.F);
                break;
            case 3:
                out[color_index] = (char) (l * 255.F);
                out[color_index + 1] = (char) (m * 255.F);
                out[color_index + 2] = (char) (VALUE * 255.F);
                break;
            case 4:
                out[color_index] = (char) (n * 255.F);
                out[color_index + 1] = (char) (l * 255.F);
                out[color_index + 2] = (char) (VALUE * 255.F);
                break;
            case 5:
                out[color_index] = (char) (VALUE * 255.F);
                out[color_index + 1] = (char) (l * 255.F);
                out[color_index + 2] = (char) (m * 255.F);
                break;
        }
    }
}

char *create_fractal() {
    std::cout << "center : " << center_x << " " << center_y << std::endl;
    std::cout << "dimension : " << r_width << " " << r_height << std::endl;
    std::cout << "max iteration : " << max_iter << std::endl;

    //calling the kernel !
    drawFractal << < nb_block, nb_threads >> >
                               (deviceImage, center_x, center_y, r_width, r_height, width, height, max_iter);

    HANDLE_ERROR(hipGetLastError());
    HANDLE_ERROR(hipDeviceSynchronize());

    size_t size = height * width * sizeof(char) * NUM_COLOR;
    HANDLE_ERROR(hipMemcpy(hostImage, deviceImage, size, hipMemcpyDeviceToHost));

    return hostImage;
}

void init_gpu(int w, int h) {
    height = h;
    width = w;

    r_height = 2.5;
    r_width = r_height * (double) width / (double) height;

    std::cout << "dimension " << r_width << " " << r_height << std::endl;

    size_t size = height * width * sizeof(char) * NUM_COLOR;

    hostImage = (char *) malloc(size);
    HANDLE_ERROR(hipMalloc(&deviceImage, size));

    nb_block = ceil((double) (height * width) / (double) NUM_THREADS_PER_BLOCK);
    nb_threads = NUM_THREADS_PER_BLOCK;
}

void delete_gpu() {
    hipFree(deviceImage);
    free(hostImage);
}

void set_center(int pos_x, int pos_y) {
    center_x = ((double) pos_x / (double) width * r_width) + center_x - r_width / 2.0;
    center_y = ((double) (height - pos_y) / (double) height * r_height) + center_y - r_height / 2.0;
}

void set_zoom_scale(double scale) {
    r_width *= scale;
    r_height *= scale;

    max_iter += 20 * ((scale < 1) ? 1 : -1);
}
